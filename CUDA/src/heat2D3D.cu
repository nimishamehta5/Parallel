
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<stdio.h>
#include<cstdio>
#include<vector>
#include<stdlib.h>
#include<string.h>
#include<cstring>
#include<sys/time.h>

using namespace std;

const int TPB=1024;
int NX;
int NY;
int NZ;
int tsteps;
float l;
int dim;
float init_t;

void init(float *T)
{
	if(dim==2)
	{
		for(int j=0;j<NY;j++)
			for(int i=0;i<NX;i++)
			{
				int index=i+j*NX;
				T[index]=init_t;
			}
	}

	else if(dim==3)
	{
		for(int k=0;k<NZ;k++)
		{
			for(int j=0;j<NY;j++)
			{
				for(int i=0;i<NX;i++)
				{
					int index=i+j*NX+k*(NX*NY);
					T[index]=init_t;
				}
			}
		}
	}
}

 void set_const(float *T, float *T_const)
 {
 	if(dim==2)
 	{
 	for(int j=0;j<NY;j++)
		for(int i=0;i<NX;i++)
		{
			int index=i+j*NX;
			if(T_const[index]!=(-1))
				T[index]=T_const[index];
		}
 	}
 	else if(dim==3)
 	{
 		for(int k=0;k<NZ;k++)
		{
			for(int j=0;j<NY;j++)
			{
				for(int i=0;i<NX;i++)
				{
					int index=i+j*NX+k*(NX*NY);
					if(T_const[index]!=(-1))
						T[index]=T_const[index];
				}
			}
		}
 	}	
 }

// __global__ void print_kernel(float *T_old, float *T_new,float *T_const_GPU,int *dim_GPU,float *l_GPU,int *NX_GPU,int *NY_GPU,int *NZ_GPU)
// {

// 		//printf("\ndim_GPU: %d, l_GPU: %f, NX_GPU: %d, NY_GPU: %d, NZ_GPU: %d\n",*dim_GPU,*l_GPU,*NX_GPU,*NY_GPU,*NZ_GPU);
// 		printf("\nT1 - %d")
	
// }

__global__ void swap_kernel(float *T_old, float *T_new,int *dim_GPU,int *NX_GPU,int *NY_GPU,int *NZ_GPU)
{

	//T1=T2
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(*dim_GPU==2)
	{
		if(index<(*NX_GPU)*(*NY_GPU))
		{
			// printf("\nT_old before swap:\n");
			// printf("index: %d, T_old[index]: %d",index,T_old[index]);

			T_old[index]=T_new[index];

			// printf("\nT_old after swap:\n");
			// printf("index: %d, T_old[index]: %d",index,T_old[index]);
		}
	}

	else if(*dim_GPU==3)
	{
		if(index<(*NX_GPU)*(*NY_GPU)*(*NZ_GPU))
		{
			// printf("\nINSIDE 3D SWAP\n");
			// printf("\nT_old before swap:\n");
			// printf("index: %d, T_old[index]: %d",index,T_old[index]);

			T_old[index]=T_new[index];

			// printf("\nT_old after swap:\n");
			// printf("index: %d, T_old[index]: %d",index,T_old[index]);
		}
	}
}

 __global__ void update_const(float *T, float *T_const_GPU, int *dim_GPU, int *NX_GPU,int *NY_GPU,int *NZ_GPU)
 {
 	if(*dim_GPU==2)
 	{
	 	int index = threadIdx.x + blockIdx.x * blockDim.x;
		 if(index<(*NX_GPU)*(*NY_GPU))
		 {
		 	//printf("index: %d, %f", index, T[index]);
		 	if(T_const_GPU[index]!=(-1))
		 	{
				T[index]=T_const_GPU[index];
				//printf("index: %d, %f ---", index, T_const_GPU[index]);
		 	}
	 	}
	}
	else if(*dim_GPU==3)
	{
		int index = threadIdx.x + blockIdx.x * blockDim.x;	
		if(index<(*NX_GPU)*(*NY_GPU)*(*NZ_GPU))
		{
		if(T_const_GPU[index]!=(-1))
			T[index]=T_const_GPU[index];
		}
	}
 }

__global__ void update(int *t_GPU,float *T_old, float *T_new,float *T_const_GPU,int *dim_GPU,float *l_GPU,int *NX_GPU,int *NY_GPU,int *NZ_GPU)
{
	int index=0,left,right,top,bott,front,back;
	// printf("\ntimestep: %d\n",(*t_GPU));
	index = threadIdx.x + blockIdx.x * blockDim.x;

	if(*dim_GPU==2)
	{
		if(index<((*NX_GPU)*(*NY_GPU)))
		{
			if(index % (*NX_GPU) == 0)
				left=index;
			else
				left=index-1;

			if((index + 1) % (*NX_GPU) == 0)
				right=index;
			else
				right=index+1;

			if(index / (*NX_GPU) == 0)
				top=index;
			else
				top=index-(*NX_GPU);

			if(((((*NX_GPU)*(*NY_GPU))-index-1) / (*NX_GPU)) == 0)
				bott=index;
			else
				bott=index+(*NX_GPU);

			//printf("index: %d, right: %f:, left: %f, top: %f, bottom: %f\n", index, T_old[right], T_old[left], T_old[top], T_old[bott]);

			T_new[index]=T_old[index] + (*l_GPU) *(T_old[left] + T_old[right] + T_old[top] + T_old[bott] - (4*T_old[index]));
		}
	}
	else if(*dim_GPU==3)
	{				
		if(index<(*NX_GPU)*(*NY_GPU)*(*NZ_GPU))
		{
			if(index % *NX_GPU == 0)
				left=index;
			else
				left=index-1;

			if((index + 1) % *NX_GPU == 0)
				right=index;
			else
				right=index+1;

			if(index / ((*NX_GPU) * (*NY_GPU)) == 0)
				front=index;
			else
				front=index-((*NX_GPU)*(*NY_GPU));

			if(( ((*NX_GPU)*(*NY_GPU)*(*NZ_GPU)) - index - 1) / ((*NX_GPU)*(*NY_GPU)) == 0)
				back=index;
			else
				back=index+((*NX_GPU)*(*NY_GPU));

			if((index%((*NX_GPU)*(*NY_GPU)))/(*NX_GPU) == 0)
				top=index;
			else
				top=index-(*NX_GPU);

			if((( ((*NX_GPU)*(*NY_GPU)*(*NZ_GPU))-index-1 ) % ((*NX_GPU)*(*NY_GPU))) / *NX_GPU == 0)
				bott=index;
			else
				bott=index+(*NX_GPU);

			T_new[index]=T_old[index] + (*l_GPU) *(T_old[left] + T_old[right] + T_old[top] + T_old[bott] + T_old[front] + T_old[back] - 6*T_old[index]);
		}
	}
}


int main(int argc,char **argv)
{
	int i,j,k,index,t,num_block,ctr=0;
	float *T_const;
	char line[500];
	char *token;
	char *filename=argv[1];
	FILE * outf=fopen("heatOutput.csv","w");

	FILE * file;
	file=fopen(filename,"r");
	while(fgets(line,sizeof(line),file))
	{
		if(line[0]!='#')
		{
			ctr++;
			if(ctr==1)
			{
				dim=line[0]-'0';
				//cout<<"\ndim: "<<dim;
			}

			if(ctr==2)
			{
				token=strtok(line,"\n");
				l=atof(token);
				//cout<<"\nl: "<<l;
			}

			if(ctr==3)
			{
				token=strtok(line,"\n");
				tsteps=atoi(token);
				//cout<<"\ntsteps: "<<tsteps;
			}

			if(ctr==4)
			{
				if(dim==2)
				{
					token=strtok(line,",");
					if(token!=NULL)
					{
						NX=atoi(token);
						//cout<<"\n2D NX: "<<NX;
						token=strtok(NULL,"\n");
					}

					if(token!=NULL)
					{
						NY=atoi(token);
						//cout<<"\n2D NY: "<<NY;
					}

						NZ=0;

					T_const=new float[NX*NY];
					for(j=0;j<NY;j++)	//Initializing T_const to -1
						for(i=0;i<NX;i++)
						{
							index=i+j*NX;
							T_const[index]=-1;
						}
				}
				else if(dim==3)
				{
					token=strtok(line,",");
					if(token!=NULL)
					{
						NX=atoi(token);
						//cout<<"\n3D NX: "<<NX;
						token=strtok(NULL,",");
					}
					if(token!=NULL)
					{
						NY=atoi(token);
						//cout<<"\n3D NY: "<<NY;
						token=strtok(NULL,"\n");
					}
					if(token!=NULL)
					{
						NZ=atoi(token);
						//cout<<"\n3D NZ: "<<NZ;
					}
					T_const=new float[NX*NY*NZ];
					for(k=0;k<NZ;k++)	//Initializing T_const to -1
					{
						for(j=0;j<NY;j++)
						{
							for(i=0;i<NX;i++)
							{
								index=i+j*NX+k*(NX*NY);
								T_const[index]=-1;
							}
						}
					}
				}
			}

			if(ctr==5)
			{
				token=strtok(line,"\n");
				init_t=atof(token);
				//cout<<"\ninit_t: "<<init_t;
			}

			if(ctr>5)
			{
				if(dim==2)
				{
					int x2,y2,w2,h2;
					float t2;

					token=strtok(line,",");
					if(token!=NULL)
						{ x2=atoi(token); token=strtok(NULL,","); }
					if(token!=NULL)
						{ y2=atoi(token); token=strtok(NULL,","); }
					if(token!=NULL)
						{ w2=atoi(token); token=strtok(NULL,","); }
					if(token!=NULL)
						{ h2=atoi(token); token=strtok(NULL,"\n"); }
					if(token!=NULL)
						{ t2=atof(token); }

					for(j=y2;j<y2+h2;j++)	//creating T_const matrix 
						for(i=x2;i<x2+w2;i++)
						{
							index=i+j*NX;
							T_const[index]=t2;
						}	

				}
				else if(dim==3)
				{
					int x3,y3,z3,w3,h3,d3;
					float t3;

					token=strtok(line,",");
					if(token!=NULL)
						{ x3=atoi(token); token=strtok(NULL,","); }
					if(token!=NULL)
						{ y3=atoi(token); token=strtok(NULL,","); }
					if(token!=NULL)
						{ z3=atoi(token); token=strtok(NULL,","); }
					if(token!=NULL)
						{ w3=atoi(token); token=strtok(NULL,","); }
					if(token!=NULL)
						{ h3=atoi(token); token=strtok(NULL,","); }
					if(token!=NULL)
						{ d3=atoi(token); token=strtok(NULL,"\n"); }
					if(token!=NULL)
						{ t3=atof(token); }

					for(k=z3;k<z3+d3;k++)	//Create const matrix
					{
						for(j=y3;j<y3+h3;j++)
						{
							for(i=x3;i<x3+w3;i++)
							{
								index=i+j*NX+k*(NX*NY);
								T_const[index]=t3;
							}
						}
					}
				}
			}
		}
	}
				

	if(dim==2)
	{
		float *T=new float[NX*NY];	 		//CPU Memory
		float *T1, *T2, *T_const_GPU;		//GPU Memory
		//float *T_temp=new float[NX*NY];
		int *dim_CPU,*NX_CPU,*NY_CPU,*NZ_CPU,*t_CPU;	//t_CPU and t_GPU created only to print timesteps from kernel
		float *l_CPU;

		dim_CPU=&dim; l_CPU=&l; NX_CPU=&NX; NY_CPU=&NY; NZ_CPU=&NZ; 

		int *dim_GPU,*NX_GPU,*NY_GPU,*NZ_GPU,*t_GPU;
		float *l_GPU;

		init(T);

		set_const(T,T_const);	//copying constants from T_const to T

		hipMalloc((void**)&T1,NX*NY*sizeof(float));	//T_old
		hipMalloc((void**)&T2,NX*NY*sizeof(float));	//T_new
		hipMalloc((void**)&T_const_GPU,NX*NY*sizeof(float)); //T_const

		hipMalloc((void**)&dim_GPU,sizeof(int));
		hipMalloc((void**)&t_GPU,sizeof(int));
		hipMalloc((void**)&l_GPU,sizeof(float));
		hipMalloc((void**)&NX_GPU,sizeof(int));
		hipMalloc((void**)&NY_GPU,sizeof(int));
		hipMalloc((void**)&NZ_GPU,sizeof(int));

		hipMemcpy(dim_GPU,dim_CPU,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(l_GPU,l_CPU,sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(NX_GPU,NX_CPU,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(NY_GPU,NY_CPU,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(NZ_GPU,NZ_CPU,sizeof(int),hipMemcpyHostToDevice);

		hipMemcpy(T1,T,NX*NY*sizeof(float),hipMemcpyHostToDevice);
		//cudaMemcpy(T2,T,NX*NY*sizeof(float),cudaMemcpyHostToDevice);
		hipMemcpy(T_const_GPU,T_const,NX*NY*sizeof(float),hipMemcpyHostToDevice);

		num_block=((NX*NY)+TPB-1)/TPB;	//TPB = Threads Per Block = 1024

		//print_kernel<<<1,1>>>(T1,T2,T_const_GPU,dim_GPU,l_GPU,NX_GPU,NY_GPU,NZ_GPU);

		//struct timeval start,end;
		//double diff;
		//gettimeofday(&start,NULL);

		for(t=0;t<tsteps;t++)
		{
			t_CPU=&t;
			hipMemcpy(t_GPU,t_CPU,sizeof(int),hipMemcpyHostToDevice);
	
			update<<<num_block,TPB>>>(t_GPU,T1,T2,T_const_GPU,dim_GPU,l_GPU,NX_GPU,NY_GPU,NZ_GPU);



			update_const<<<num_block,TPB>>>(T2,T_const_GPU,dim_GPU,NX_GPU,NY_GPU,NZ_GPU);
			
			// cudaMemcpy(T_temp, T2, NX*NY*sizeof(float), cudaMemcpyDeviceToHost);	//SWAP IN CPU IMPLEMENTATION
			// cout<<"\nT_TEMP:\n";
			// for(int j=0;j<NY;j++)
			// {
			// 	for(int i=0;i<NX;i++)
			// 	{
			// 		int index=i+j*NX;
			// 		cout<<" "<<T_temp[index]<<" ";
			// 	}
			// 	cout<<"\n";
			// }
			//cudaMemcpy(T1, T_temp, NX*NY*sizeof(float), cudaMemcpyHostToDevice);
			//update_const<<<num_block,TPB>>>(T1,T_const_GPU,dim_GPU);
			

			swap_kernel<<<num_block,TPB>>>(T1,T2,dim_GPU,NX_GPU,NY_GPU,NZ_GPU);	//KERNEL SWAP IMPLEMENTATION


			//T1=T2;	//SIMPLE(WRONG?) SWAP IMPLEMENTATION

			//DOUBLE UPDATE CALL IMPLEMENTATION
		}

	//	gettimeofday(&end,NULL);
	//	diff=(end.tv_sec - start.tv_sec)*1000.0;	//sec to ms conversion

	//	cout<<"\n2D TIME: "<<diff<<"\n";

		// if(tsteps%2==0)	//FOR DOUBLE UPDATE CALL IMPLEMENTATION
		// {
			hipMemcpy(T,T1,NX*NY*sizeof(float),hipMemcpyDeviceToHost);
		// }
		// else
		// {
		// 	cudaMemcpy(T,T1,NX*NY*sizeof(float),cudaMemcpyDeviceToHost);
		// }
		
		//cout<<"\nFINAL:\n";

		for(int j=0;j<NY;j++)
		{
			for(int i=0;i<NX;i++)
			{
				index=i+j*NX;
				if((index+1) % NX == 0)
				{
					fprintf(outf,"%f",T[index]);
					printf("%f",T[index]);
				}
				else
				{
					fprintf(outf,"%f, ",T[index]);
					printf("%f",T[index]);
				}
			}
			fprintf(outf,"\n");
			printf("\n");
		}

		delete T;
		delete T_const;

		hipFree(T1);
		hipFree(T2);
		hipFree(T_const_GPU);
		hipFree(l_GPU);
		hipFree(dim_GPU);
		hipFree(NX_GPU);
		hipFree(NY_GPU);
		hipFree(NZ_GPU);
		hipFree(t_GPU);
	}

	else if(dim==3)
	{
		float *T=new float[NX*NY*NZ];	 		//CPU Memory
		float *T1, *T2, *T_const_GPU;		//GPU Memory
		//float *T_temp=new float[NX*NY];
		int *dim_CPU,*NX_CPU,*NY_CPU,*NZ_CPU,*t_CPU;	//t_CPU and t_GPU created only to print timesteps from kernel
		float *l_CPU;

		dim_CPU=&dim; l_CPU=&l; NX_CPU=&NX; NY_CPU=&NY; NZ_CPU=&NZ; 

		int *dim_GPU,*NX_GPU,*NY_GPU,*NZ_GPU,*t_GPU;
		float *l_GPU;

		init(T);

		set_const(T,T_const);	//copying constants from T_const to T

		hipMalloc((void**)&T1,NX*NY*NZ*sizeof(float));	//T_old
		hipMalloc((void**)&T2,NX*NY*NZ*sizeof(float));	//T_new
		hipMalloc((void**)&T_const_GPU,NX*NY*NZ*sizeof(float));

		hipMalloc((void**)&dim_GPU,sizeof(int));
		hipMalloc((void**)&l_GPU,sizeof(float));
		hipMalloc((void**)&NX_GPU,sizeof(int));
		hipMalloc((void**)&NY_GPU,sizeof(int));
		hipMalloc((void**)&NZ_GPU,sizeof(int));
		hipMalloc((void**)&t_GPU,sizeof(int));

		hipMemcpy(dim_GPU,dim_CPU,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(l_GPU,l_CPU,sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(NX_GPU,NX_CPU,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(NY_GPU,NY_CPU,sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(NZ_GPU,NZ_CPU,sizeof(int),hipMemcpyHostToDevice);

		hipMemcpy(T1,T,NX*NY*NZ*sizeof(float),hipMemcpyHostToDevice);
		//cudaMemcpy(T2,T,NX*NY*NZ*sizeof(float),cudaMemcpyHostToDevice);
		hipMemcpy(T_const_GPU,T_const,NX*NY*NZ*sizeof(float),hipMemcpyHostToDevice);

		num_block=((NX*NY*NZ)+TPB-1)/TPB;	//TPB = Threads Per Block = 1024

	//	struct timeval start,end;
	//	double diff;
	//	gettimeofday(&start,NULL);

		for(t=0;t<tsteps;t++)
		{
			t_CPU=&t;
			hipMemcpy(t_GPU,t_CPU,sizeof(int),hipMemcpyHostToDevice);

			update<<<num_block,TPB>>>(t_GPU,T1,T2,T_const_GPU,dim_GPU,l_GPU,NX_GPU,NY_GPU,NZ_GPU);
			update_const<<<num_block,TPB>>>(T2,T_const_GPU,dim_GPU,NX_GPU,NY_GPU,NZ_GPU);

			//update_const<<<num_block,TPB>>>(T1,T_const_GPU,dim_GPU);
			//swap
			//T1=T2;

			//swap_kernel<<<num_block,TPB>>>(T1,T2,dim_GPU,NX_GPU,NY_GPU,NZ_GPU);

			// update<<<num_block,TPB>>>(t_GPU,T2,T1,T_const_GPU,dim_GPU,l_GPU,NX_GPU,NY_GPU,NZ_GPU);
			// update_const<<<num_block,TPB>>>(T1,T_const_GPU,dim_GPU);
			// update_const<<<num_block,TPB>>>(T2,T_const_GPU,dim_GPU);
		}

	//	gettimeofday(&end,NULL);
	//	diff=(end.tv_sec - start.tv_sec)*1000.0;	//sec to ms conversion

	//	cout<<"\n3D TIME: "<<diff<<"\n";

		// if(tsteps%2==0)
		// {
			hipMemcpy(T,T2,NX*NY*NZ*sizeof(float),hipMemcpyDeviceToHost);
		// }
		// else
		// {
		// 	cudaMemcpy(T,T1,NX*NY*NZ*sizeof(float),cudaMemcpyDeviceToHost);
		// }
		
		/*for(int i=0;i<NX*NY*NZ;i++)
		{
		if(i%NX==0)
			fprintf(outf,"\n");
		if(i%(NX*NY)==0)
			fprintf(outf,"\n");
		if(i%NX==NX-1)
			fprintf(outf,"%f ",T[i]);
		else
			fprintf(outf,"%f, ",T[i]);
		}*/

		for(int k=0;k<NZ;k++)
		{
			for(int j=0;j<NY;j++)
			{
				for(int i=0;i<NX;i++)
				{
					index=i+j*NX+k*(NX*NY);
					if((index+1) % NX == 0)
					{
						fprintf(outf,"%f",T[index]);
						printf("%f",T[index]);
					}
					else
					{
						fprintf(outf,"%f, ",T[index]);
						printf("%f",T[index]);
					}
				}
				fprintf(outf,"\n");
				printf("\n");
			}
			fprintf(outf,"\n");
			printf("\n");
		}

		delete T;
		delete T_const;

		hipFree(T1);
		hipFree(T2);
		hipFree(T_const_GPU);
		hipFree(l_GPU);
		hipFree(dim_GPU);
		hipFree(NX_GPU);
		hipFree(NY_GPU);
		hipFree(NZ_GPU);
		hipFree(t_GPU);
	}

		return 0;
}

	//GPU JOB: qsub -I -q coc-ice -l nodes=1:ppn=2:gpus=1,walltime=2:00:00,pmem=2gb
	//CPU JOB: qsub -I -q coc-ice -l nodes=1:ppn=12,walltime=2:00:00,pmem=2gb