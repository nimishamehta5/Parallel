#include "hip/hip_runtime.h"


#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include "complex.h"
#include "input_image.h"
#include <bits/stdc++.h>
#include <string>
#include "complex.cc"
#include "input_image.cc"
#include <chrono>

//const float PI = 3.14159265358979f;
//threadsperblock 1024
__global__ void compute_gpu_dft(Complex *arr_old, Complex *arr_new, int *d_num_blocks, float *d_PI_val)
{
    int num_blocks = *d_num_blocks;
    float PI_val = *d_PI_val;
	int row_id = blockIdx.x/num_blocks;
	int row_elements = blockDim.x*num_blocks;
	int x_value = (blockIdx.x%num_blocks)*blockDim.x+threadIdx.x;
	int idx = x_value*row_elements + row_id;

	float rl = 0;
	float im = 0;

	for( int j = 0; j < row_elements; j++)
	{
	    float sinval, cosval;
	    //coef[i*numx+j] = Complex(cos(-2.*PI*j*x_value/row_elements), sin(-2.*PI*j*x_value/row_elements));
	    __sincosf ( -2.*PI_val*j*x_value/row_elements, &sinval, &cosval );
	    float old_real = arr_old[j+row_id*row_elements].real;
	    float old_imag = arr_old[j+row_id*row_elements].imag;

//		rl += arr_old[j+row_id*row_elements].real*coef[x_value+j*row_elements].real - arr_old[j+row_id*row_elements].imag*coef[ x_value+j*row_elements].imag;
//		im += arr_old[j+row_id*row_elements].real*coef[ x_value+j*row_elements].imag + arr_old[j+row_id*row_elements].imag*coef[ x_value+j*row_elements].real;

		rl += old_real*cosval - old_imag*sinval;
		im += old_real*sinval + old_imag*cosval;

	}
	arr_new[idx].real = rl;
	arr_new[idx].imag = im;
}


int main( int argc, char ** argv)
{

int numberofblocks=2;
//auto start = std::chrono::system_clock::now();
//FILE *fp, *fp1;
//fp1 = std::fopen(argv[3], "w");
//fp = std::fopen("heatOutput.txt", "w");
Complex *arr,*arr_old;
//Complex *W;
InputImage input_file(argv[2]);
int numx = input_file.get_width();
int T_P_B = input_file.get_height()/numberofblocks;
Complex *freq;
int sizearr = numx*numx*sizeof(Complex);
freq=(Complex*)malloc(sizearr);
Complex *final_freq=(Complex*)malloc(sizearr);
Complex *coef = (Complex*)malloc(sizearr);
freq=input_file.get_image_data();

//auto end = std::chrono::system_clock::now();
  //  auto elapsed1 = end - start;

int numberofrows = input_file.get_height();





//cpu allocation



//char * cstr = new char [argv[1].length()+1];
//std::strcpy (cstr, argv[1].c_str());

//std::string str ="Tower1024.txt";
//char c[20] = "Tower1024.txt";
//InputImage input_file("Tower1024.txt");


// freq[0].real = 0;
// freq[1].real = 0;
// freq[2].real = 1;
// freq[3].real = 0;

// freq[0].imag = -1;
// freq[1].imag = 2;
// freq[2].imag = 0;
// freq[3].imag = 0;


//for(int i = 0; i < numx; i++)
//{
//	for( int j = 0; j < numx; j++)
//	{
//		coef[i*numx+j] = Complex(cos(-2.*PI*i*j/numx), sin(-2.*PI*i*j/numx));
//	}
//}

// std::cout << "printing the coeficient array" << std::endl;
// for(int i = 0; i < numx; i++)
// {
// 	for( int j = 0; j < numx; j++)
// 	{
// 		std::cout << coef[i*numx+j].real << " " << coef[i*numx+j].imag << std::endl;
// 	}
// }

// //gpu allocation
int *d_numblocks;float *d_pi;
int sizeint = sizeof(int);
int sizefloat = sizeof(float);
hipMalloc((void**)&arr,sizearr);
hipMalloc((void**)&arr_old,sizearr);
hipMalloc((void**)&d_numblocks,sizeint);
hipMalloc((void**)&d_pi,sizefloat);

//hipMalloc((void**)&W,sizearr);



// for(int i = 0; i < numx; i++)
// {
// 	for( int j = 0; j < numx; j++)
// 	{
// 		std::cout << freq[i*numx+j].real << " " << freq[i*numx+j].imag << std::endl;
// 	}
// }

//host to device
//_host__ ​ __device__ ​hipError_t hipMemcpyAsync ( arr_old, freq, sizearr, hipMemcpyHostToDevice, hipStream_t stream = 0 )
hipMemcpy(arr_old,freq,sizearr,hipMemcpyHostToDevice);
hipMemcpy(d_pi,&PI,sizefloat,hipMemcpyHostToDevice);
hipMemcpy(d_numblocks, &numberofblocks,sizeint,hipMemcpyHostToDevice);
//hipMemcpy(arr,final_freq,sizearr,hipMemcpyHostToDevice);
//hipMemcpy(W,coef,sizearr,hipMemcpyHostToDevice);


  //std::cout << elapsed.count() << '\n';
  //auto elapsed = std::chrono::duration_cast<std::chrono::seconds>(end - start);


	//printf("Printing I %d", i);
	compute_gpu_dft<<<numx*numberofrows / T_P_B , T_P_B>>>(arr_old,arr, d_numblocks,d_pi);



// hipMemcpy(freq,arr,sizearr,hipMemcpyDeviceToHost);
// for(int i = 0; i < numx; i++)
// {
// 	for( int j = 0; j < numx; j++)
// 	{
// 		std::cout << freq[i*numx+j].real << " " << freq[i*numx+j].imag << std::endl;
// 	}
// }


	compute_gpu_dft<<<numx*numberofrows / T_P_B, T_P_B>>>(arr,arr_old, d_numblocks,d_pi);





//hipMemcpy(freq,arr,sizearr,hipMemcpyDeviceToHost);
hipMemcpy(final_freq,arr_old,sizearr,hipMemcpyDeviceToHost);

	// for(int i = 0; i < numx; i++)
	// {
	// 	for(int j = 0; j < numx;j++)
	// 	{
	// 		std::cout << final_freq[i*numx+j].real << std::endl;
	// 	}
	// }

	//end = std::chrono::system_clock::now();
	//auto elapsed2 = end - start;
  //std::cout << elapsed.count() << '\n';
  //auto elapsed = std::chrono::duration_cast<std::chrono::seconds>(end - start);
  //std::cout << "computation taken in nano seconds: " << (elapsed2-elapsed1).count() << '\n';

    //	for(int i = 0; i < numx; i++)
    //	{
    //		for(int j = 0; j < numx;j++)
    //		{
    //			//std::cout << final_freq[i*numx+j].real/(numx*numx) << std::endl;
    //			fprintf(fp1,"(%f,",final_freq[i*numx+j].real );
    //			fprintf(fp1,"%f) ",final_freq[i*numx+j].imag );
    //		}
    //		fprintf(fp1, "\n");
    //	}

    input_file.save_image_data(argv[3], final_freq, numx, numx);
	// int i,j,rhs,lhs;
	// for(i=0;i<numx;i++)
	// {
	// 	for(j=0;j<numx;j++)
	// 	{
	// 		rhs=i+j*numx;
	// 		lhs=j+i*numx;
	// 		freq[lhs]=final_freq[rhs];
	// 	}
	// }

	// for(int i = 0; i < numx; i++)
	// {
	// 	for( int j = 0; j < numx; j++)
	// 	{
	// 		coef[i*numx+j] = Complex(cos(2.*PI*i*j/numx), sin(2.*PI*i*j/numx));
	// 	}
	// }

	// // std::cout << "printing the coeficient array" << std::endl;
	// // for(int i = 0; i < numx; i++)
	// // {
	// // 	for( int j = 0; j < numx; j++)
	// // 	{
	// // 		std::cout << coef[i*numx+j].real << " " << coef[i*numx+j].imag << std::endl;
	// // 	}
	// // }

	// hipMemcpy(arr_old,freq,sizearr,hipMemcpyHostToDevice);
	// hipMemcpy(arr,final_freq,sizearr,hipMemcpyHostToDevice);
	// hipMemcpy(W,coef,sizearr,hipMemcpyHostToDevice);
	// for( int i = 0; i < numx; i++)
	// {

	// 	//printf("Printing I %d", i);
	// 	compute_gpu_dft<<<numx / T_P_B, T_P_B>>>(arr_old,arr, W , i);
	// }

	// hipDeviceSynchronize();

	// hipMemcpy(freq,arr,sizearr,hipMemcpyDeviceToHost);
	// // for(int i = 0; i < numx; i++)
	// // {
	// // 	for( int j = 0; j < numx; j++)
	// // 	{
	// // 		std::cout << freq[i*numx+j].real << " " << freq[i*numx+j].imag << std::endl;
	// // 	}
	// // }

	// for( int i = 0; i < numx; i++)
	// {
	// 	compute_gpu_dft<<<numx / T_P_B, T_P_B>>>(arr,arr_old, W,i);
	// }

	// hipMemcpy(freq,arr,sizearr,hipMemcpyDeviceToHost);
	// hipMemcpy(final_freq,arr_old,sizearr,hipMemcpyDeviceToHost);

	// for(i=0;i<numx;i++)
	// {
	// 	for(j=0;j<numx;j++)
	// 	{
	// 		rhs=i+j*numx;
	// 		lhs=j+i*numx;
	// 		freq[lhs]=final_freq[rhs];
	// 	}
	// }

	// for(int i = 0; i < numx; i++)
	// {
	// 	for(int j = 0; j < numx;j++)
	// 	{
	// 		//std::cout << final_freq[i*numx+j].real/(numx*numx) << std::endl;
	// 		fprintf(fp,"%f ",freq[i*numx+j].real/(numx*numx) );
	// 		//fprintf(fp,"%f ",final_freq[i*numx+j].imag/(numx*numx) );
	// 	}
	// 	fprintf(fp, "\n");
	// }

  hipFree(final_freq);
  //hipFree(W);
  hipFree(freq);
  hipFree(d_numblocks);
  hipFree(d_pi);

  //end = std::chrono::system_clock::now();
  //auto elapsed = end - start;
  //std::cout << elapsed.count() << '\n';
  //auto elapsed = std::chrono::duration_cast<std::chrono::seconds>(end - start);
  //std::cout << "Time taken in nano seconds: " << elapsed.count() << '\n';
  //fclose(fp);
  //fclose(fp1);
  return 0;
}
